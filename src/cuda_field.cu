
#include <hip/hip_runtime.h>

typedef long long LL;

__device__ 
int cuda_field_modulus;

__device__
int inverse(int a, int p){
	return a == 1 ? 1 : ((LL)(a-inverse(p%a, a))*p+1)/a;
}

__device__
void cuda_field_init(int m){
	cuda_field_modulus = m;
}

struct cuda_field_element {
	__device__
	cuda_field_element() : value(0) {}
	
	__device__
	cuda_field_element(const cuda_field_element &v) : value(v.value) {}

	__device__
	cuda_field_element(int v) : value(v) { value %= cuda_field_modulus; }


	__device__
	cuda_field_element& operator=(const cuda_field_element &v){ value = v.value; return *this; }

	__device__
	cuda_field_element& operator+=(const cuda_field_element &v){
		value += v.value; if(value >= cuda_field_modulus) value -= cuda_field_modulus; return *this;
	}

	__device__
	cuda_field_element& operator-=(const cuda_field_element &v){
		value -= v.value; if(value < 0) value += cuda_field_modulus; return *this;
	}

	__device__
	cuda_field_element& operator*=(const cuda_field_element &v){ value *= v.value; value %= cuda_field_modulus; return *this; }
	

	__device__
	cuda_field_element operator+(const cuda_field_element &v) const { return cuda_field_element(*this) += v; }

	__device__
	cuda_field_element operator-(const cuda_field_element &v) const { return cuda_field_element(*this) -= v; }

	__device__
	cuda_field_element operator*(const cuda_field_element &v) const { return cuda_field_element(*this) *= v; }


	__device__
	cuda_field_element operator-() const { return -value; }

	__device__
	cuda_field_element operator~() const { return inverse(value, cuda_field_modulus); }


	__device__
	bool operator==(const cuda_field_element &v) const { return value == v.value; }

	__device__
	bool operator!=(const cuda_field_element &v) const { return value != v.value; }


	__device__
	int get_value() const { return value; }

private:
	int value;
};
