#include "hip/hip_runtime.h"
#include "cuda_field.cu"

extern "C" {

__global__
void field_init(int m){
	cuda_field_init(m);
}

__global__
void make_unit(cuda_field_element* B, int n){
	int thidX = (blockIdx.x * blockDim.x) + threadIdx.x;
	int thidY = (blockIdx.y * blockDim.y) + threadIdx.y;
	if(thidX < n && thidY < n){
		if(thidX == thidY){
			B[thidX*n+thidY] = 1;
		}else{
			B[thidX*n+thidY] = 0;
		}
	}
}

__global__
void find_nonzero(cuda_field_element* A, int n, int i, int* k){
	int thidX = (blockIdx.x * blockDim.x) + threadIdx.x;
	if(i < thidX && thidX < n)
		if(A[thidX*n+i] != 0)
			*k = thidX;
}

__global__
void swap(cuda_field_element* M, int n, int i, int k){
	int thidX = (blockIdx.x * blockDim.x) + threadIdx.x;
	if(thidX < n){
		cuda_field_element v = M[i*n+thidX];
		M[i*n+thidX] = M[k*n+thidX];
		M[k*n+thidX] = v;
	}
}

__global__
void fix_row(cuda_field_element* M, int n, int i, cuda_field_element mul){
	int thidX = (blockIdx.x * blockDim.x) + threadIdx.x;
	if(thidX < n){
		M[i*n+thidX] = M[i*n+thidX] * mul;
	}
}

__global__
void update_column(cuda_field_element* A, cuda_field_element* i_th_column, int n, int i){
	int thidX = (blockIdx.x * blockDim.x) + threadIdx.x;
	if(thidX < n){
		i_th_column[thidX] = A[thidX*n + i];
	}
}

__global__
void fix_column(cuda_field_element* M, cuda_field_element* i_th_column, int n, int i){
	int thidX = (blockIdx.x * blockDim.x) + threadIdx.x;
	int thidY = (blockIdx.y * blockDim.y) + threadIdx.y;
	__shared__ cuda_field_element P[32];
	if(thidY > i && thidY < n){
		P[threadIdx.x] = M[(i-1)*n+thidX];
		M[thidY*n+thidX] -= i_th_column[thidY]*P[threadIdx.x];
	}
}

}

