#include "hip/hip_runtime.h"
#include "cuda_field.cu"

extern "C" {

__global__
void field_init(int m){
	cuda_field_init(m);
}

__global__
void make_unit(cuda_field_element* M, int n){
	int thidX = (blockIdx.x * blockDim.x) + threadIdx.x;
	int thidY = (blockIdx.y * blockDim.y) + threadIdx.y;
	if(thidX < n && thidY < n){
		if(thidX == thidY){
			M[thidX*n+thidY] = 1;
		}else{
			M[thidX*n+thidY] = 0;
		}
	}
}

__global__
void find_nonzero(cuda_field_element* M, int n, int i, int* k){
	int thidX = (blockIdx.x * blockDim.x) + threadIdx.x;
	if(i < thidX && thidX < n)
		if(M[thidX*n+i] != 0)
			*k = thidX;
}

__global__
void swap(cuda_field_element* M, int n, int i, int k){
	int thidX = (blockIdx.x * blockDim.x) + threadIdx.x;
	if(thidX < n){
		cuda_field_element v = M[i*n+thidX];
		M[i*n+thidX] = M[k*n+thidX];
		M[k*n+thidX] = v;
	}
}

__global__
void fix_row(cuda_field_element* M, int n, int i, cuda_field_element mul){
	//int thidX = (blockIdx.x * blockDim.x) + threadIdx.x;
}

__global__
void fix_column(cuda_field_element* M, int n, int i){
	//int thidX = (blockIdx.x * blockDim.x) + threadIdx.x;
	//int thidY = (blockIdx.y * blockDim.y) + threadIdx.y;
}

}
