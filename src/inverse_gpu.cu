#include "hip/hip_runtime.h"
#include<cstdio>

extern "C" {
__global__
void init(int* B, int n){
    int thidX = (blockIdx.x * blockDim.x) + threadIdx.x;
    int thidY = (blockIdx.y * blockDim.y) + threadIdx.y;
   
}

//P := dev_to_process
__global__
void swap(int* P, int n, int row_id, int row_to_swap){
    int thidX = (blockIdx.x * blockDim.x) + threadIdx.x;
   
}

__global__
void fixRow(int* P, int* A, int n, int row_id, Value inv){
    int thidX = (blockIdx.x * blockDim.x) + threadIdx.x;
   
}

__global__
void fixColumn(int* P, int* A, int n, int column_id, Value inv){
    int thidX = (blockIdx.x * blockDim.x) + threadIdx.x;
    int thidY = (blockIdx.y * blockDim.y) + threadIdx.y;
   
}
}
